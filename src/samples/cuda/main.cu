#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>

//#include <helper_functions.h>
//#include <helper_cuda.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif
__global__ void testKernel(int val)
{
	printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y*gridDim.x+blockIdx.x, \
		threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x, val);
}

int main(int argc, char **argv)
{
	int devID;
	hipDeviceProp_t props;

//	devID = findCudaDevice(argc, (const char **) argv);

	//checkCudaErrors(cudaGetDevice(&devID));
	//checkCudaErrors(cudaGetDeviceProperties(&props, devID));

	//printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

	printf("printf() is called. Output:\n\n");

	dim3 dimGrid(2,2);
	dim3 dimBlock(2,2,2);

	testKernel<<<dimGrid, dimBlock>>>(10);
	hipDeviceSynchronize();

	hipDeviceReset();

	return EXIT_SUCCESS;
}
